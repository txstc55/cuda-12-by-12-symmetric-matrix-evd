#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "householder_n_12.cuh"
// #include "qr_n_12.cuh"
#include "qr_n_12_tri_diagonal.cuh"
#include "evd_12.cuh"
#include <vector>

// Error checking macro
#define CHECK_CUDA(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

#define CHECK_CUSOLVER(call) { \
    const hipsolverStatus_t error = call; \
    if (error != HIPSOLVER_STATUS_SUCCESS) { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("CUSOLVER error code: %d\n", error); \
        exit(1); \
    } \
}

__global__ void generateSymmetricMatrices(double *d_A, int n, unsigned long long seed) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n) return;
    int N = 12;

    // Initialize the random number generator
    hiprandState_t state;
    hiprand_init(seed, tid, 0, &state);

    // Generate the upper triangle of the matrix
    for (int row = 0; row < N; ++row) {
        for (int col = row; col < N; ++col) {
            double randomValue = hiprand_uniform_double(&state) * 2000 - 1000.0;
            d_A[tid * N * N + row * N + col] = randomValue; // Upper triangle
            d_A[tid * N * N + col * N + row] = randomValue; // Mirror to lower triangle
        }
    }

}

__global__ void evd(double* d_A, double* d_e, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n) return;
    evd_12(d_A + tid * 144, d_e + tid * 12);
}

int main() {
    // ===============================================
    // Setup
    // ===============================================
    hipsolverHandle_t cusolverH = NULL;
    hipsolverSyevjInfo_t syevj_params = NULL;
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // Compute eigenvalues and eigenvectors
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    int n = 12; // size of each matrix
    int lda = n;
    int batchSize = 1000000;
    double *d_A = NULL; // Device matrix
    double *d_W = NULL; // Device eigenvalues
    int *d_info = NULL; // info on device
    double *d_work = NULL; // workspace on device
    int lwork = 0; // workspace size
    int info_gpu = 0; // info on host
    hipEvent_t start, stop;
    float milliseconds = 0;
    unsigned int selected_index = 0;
    std::vector<double> selected_matrix;
    selected_matrix.resize(n * n);
    std::vector<double> selected_eigen_vectors;
    selected_eigen_vectors.resize(n * n);
    std::vector<double> selected_eigen_values;
    selected_eigen_values.resize(n);


    // Initialize cuSolver
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH));
    // Allocate memory on device
    CHECK_CUDA(hipMalloc((void**)&d_A, sizeof(double) * lda * n * batchSize));
    CHECK_CUDA(hipMalloc((void**)&d_W, sizeof(double) * n * batchSize));
    CHECK_CUDA(hipMalloc((void**)&d_info, sizeof(int) * batchSize));
    // Setup the execution configuration
    int threadsPerBlock = 32;
    int blocksPerGrid = (batchSize + threadsPerBlock - 1) / threadsPerBlock;
    std::vector<double> eigenVectors;
    std::vector<double> eigenValues;
    eigenVectors.resize(n * n * batchSize);
    eigenValues.resize(n * batchSize);
    double maximumError = 0.0;


    // ===============================================
    // Generate random symmetric matrices
    // ===============================================
    // Seed for the random number generator
    unsigned long long seed = 12887265;
    // Launch the kernel to generate random symmetric matrices
    generateSymmetricMatrices<<<blocksPerGrid, threadsPerBlock>>>(d_A, batchSize, seed);
    hipDeviceSynchronize();
    std::vector<double> manualMat = {0.6392813911953831, 0.0000000000000110, 0.0000000418986418, -0.4261878281680344, 0.0000000414003598, 0.0966623964244656, -0.4261874926604000, -0.0000000254645934, -0.0966624262315118, 0.2130939296330515, -0.0000000159357774, -0.0000000120915957, 
0.0000000000000110, 0.6392813911953826, 0.0000000410945048, 0.0000000415927983, -0.4261878289721961, 0.0966623393070661, -0.0000000156299344, -0.4261874507617329, 0.0000000021435920, -0.0000000259628750, 0.2130938885385465, -0.0966623825451630, 
0.0000000418986418, 0.0000000410945048, 0.7952742405529198, 0.0593304795308058, 0.0593304447832330, -0.5821807906743447, -0.0593305140077440, 0.0000000013157172, -0.4261874504153784, -0.0000000074217036, -0.0593304871934551, 0.2130940005368032, 
-0.4261878281680344, 0.0000000415927983, 0.0593304795308058, 0.5821809653294843, 0.1559928936811963, -0.1559929891040656, 0.0571008454968625, -0.0966624099369389, 0.0966624705140362, -0.2130939826583125, -0.0593305253370558, 0.0000000390592235, 
0.0000000414003598, -0.4261878289721961, 0.0593304447832330, 0.1559928936811963, 0.5821807799912386, -0.1559928964349136, -0.0593305026702667, 0.2130938576918510, -0.0000000020118598, -0.0966624324112894, -0.3690868087108939, 0.0966624536635404, 
0.0966623964244656, 0.0966623393070661, -0.5821807906743447, -0.1559929891040656, -0.1559928964349136, 0.5821809708369222, 0.0593305413217186, -0.0000000011010955, 0.2130938538861867, 0.0000000513578813, 0.0593305582289431, -0.2130940340487645, 
-0.4261874926604000, -0.0000000156299344, -0.0593305140077440, 0.0571008454968625, -0.0593305026702667, 0.0593305413217186, 0.5821805568743379, -0.0000000034593112, -0.0000000003463557, -0.2130939097108004, 0.0593305217595123, -0.0000000269676190, 
-0.0000000254645934, -0.4261874507617329, 0.0000000013157172, -0.0966624099369389, 0.2130938576918510, -0.0000000011010955, -0.0000000034593112, 0.4261875257533216, 0.0000000000000000, 0.0966624388608435, -0.2130939326834397, -0.0000000002146217, 
-0.0966624262315118, 0.0000000021435920, -0.4261874504153784, 0.0966624705140362, -0.0000000020118598, 0.2130938538861867, -0.0000000003463557, 0.0000000000000000, 0.4261875257533215, -0.0000000439361688, -0.0000000001317322, -0.2130939292241300, 
0.2130939296330515, -0.0000000259628750, -0.0000000074217036, -0.2130939826583125, -0.0966624324112894, 0.0000000513578813, -0.2130939097108004, 0.0966624388608435, -0.0000000439361688, 0.2130939627360615, 0.0000000195133208, -0.0000000000000089, 
-0.0000000159357774, 0.2130938885385465, -0.0593304871934551, -0.0593305253370558, -0.3690868087108939, 0.0593305582289431, 0.0593305217595123, -0.2130939326834397, -0.0000000001317322, 0.0000000195133208, 0.3690868528557871, -0.0000000709037557, 
-0.0000000120915957, -0.0966623825451630, 0.2130940005368032, 0.0000000390592235, 0.0966624536635404, -0.2130940340487645, -0.0000000269676190, -0.0000000002146217, -0.2130939292241300, -0.0000000000000089, -0.0000000709037557, 0.2130939627360913, };
    
    for (int i = 0; i < 1; i++){
    hipMemcpy(d_A + 144 * i, manualMat.data(), 144 * sizeof(double), hipMemcpyHostToDevice);
    }


    // ===============================================
    // check and save the matrices
    // ===============================================
    std::vector<double> As;
    As.resize(n * n * batchSize);
    hipMemcpy(As.data(), d_A, sizeof(double) * n * n * batchSize, hipMemcpyDeviceToHost);
    printf("Random matrix generated and stored on host and device\n");
    hipMemcpy(selected_matrix.data(), d_A + selected_index * n * n, sizeof(double) * n * n, hipMemcpyDeviceToHost);

    printf("The selected matrix:\n");
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            printf("%lf, ", selected_matrix[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");


    // ===============================================
    // Eigen value decomposition with our code
    // ===============================================
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    evd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_W, batchSize);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time for 1 execution of our eigen value decomposition: %f ms\n", milliseconds);

    // ===============================================
    // check the error for our solution
    // ===============================================
    hipMemcpy(eigenVectors.data(), d_A, sizeof(double) * n * n * batchSize, hipMemcpyDeviceToHost);
    hipMemcpy(eigenValues.data(), d_W, sizeof(double) * n * batchSize, hipMemcpyDeviceToHost);

    for (int i = 0; i < batchSize; i++){
        double temp[n * n];
        double eigenVector[n * n];
        double eigenValue[n];
        for (int j = 0; j < n * n; j++){
            eigenVector[j] = eigenVectors[i * n * n + j];
        }
        for (int j = 0; j < n; j++){
            eigenValue[j] = eigenValues[i * n + j];
        }

        // now perform U^T * V * U
        for (int j = 0; j < n * n; j++){
            temp[j] = 0;
        }
        for (int j = 0; j < n; j++){
            for (int k = 0; k < n; k++){
                for (int l = 0; l < n; l++){
                    temp[j * n + k] += eigenVector[j * n + l] * eigenValue[l] * eigenVector[k * n + l];
                    if (temp != temp){
                        printf("NAN detected in eigen vector or eigen values\n");
                        exit(1);
                    }
                }
            }
        }
        double diff = 0;
        for (int j = 0; j < n * n; j++){
            diff += abs(temp[j] - As[i * n * n + j]);
        }
        if (diff > maximumError){
            maximumError = diff;
        }
    }
    printf("Maximum error: %lf\n", maximumError);
    maximumError = 0.0;
    

    hipMemcpy(selected_eigen_vectors.data(), d_A + selected_index * n * n, sizeof(double) * n * n, hipMemcpyDeviceToHost);
    hipMemcpy(selected_eigen_values.data(), d_W + selected_index * n, sizeof(double) * n, hipMemcpyDeviceToHost);
    printf("The selected matrix's eigen vectors using our method:\n");
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            printf("%lf, ", selected_eigen_vectors[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
    printf("The selected matrix's eigen values using our method:\n");
    for (int i = 0; i < n; i++){
        printf("%lf, ", selected_eigen_values[i]);
    }
    printf("\n\n");

    // ===============================================
    // setting up Eigen value decomposition with cuSolver
    // ===============================================
    // first we copy back the original matrix
    hipMemcpy(d_A, As.data(), sizeof(double) * n * n * batchSize, hipMemcpyHostToDevice);

    // Create syevj parameters
    CHECK_CUSOLVER(hipsolverDnCreateSyevjInfo(&syevj_params));

    // Set syevj parameters
    // For example, setting the tolerance and the maximum number of sweeps:
    CHECK_CUSOLVER(hipsolverDnXsyevjSetTolerance(syevj_params, 1e-6));
    CHECK_CUSOLVER(hipsolverDnXsyevjSetMaxSweeps(syevj_params, 100));

    // Query working space of syevjBatched
    CHECK_CUSOLVER(hipsolverDnDsyevjBatched_bufferSize(
        cusolverH,
        jobz,
        uplo,
        n,
        d_A,
        lda,
        d_W,
        &lwork,
        syevj_params,
        batchSize));

    // Allocate workspace for device
    CHECK_CUDA(hipMalloc((void**)&d_work, sizeof(double) * lwork));
    hipDeviceSynchronize();


    // ===============================================
    // cusolver execute
    // ===============================================
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // Execute the eigendecomposition
    hipsolverDnDsyevjBatched(
        cusolverH,
        jobz,
        uplo,
        n,
        d_A,
        lda,
        d_W,
        d_work,
        lwork,
        d_info,
        syevj_params,
        batchSize);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time for 1 execution of cusolver eigen value decomposition: %f ms\n", milliseconds);
    // Synchronize and check for errors
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipMemcpy(&info_gpu, d_info, sizeof(int), hipMemcpyDeviceToHost));

    if (info_gpu == 0) {
        printf("Batched syevj execution successful!\n");
    } else {
        printf("Batched syevj execution failed: Info = %d\n", info_gpu);
    }

    // ===============================================
    // check the error for cuSolver solution
    // ===============================================

    hipMemcpy(eigenVectors.data(), d_A, sizeof(double) * n * n * batchSize, hipMemcpyDeviceToHost);
    hipMemcpy(eigenValues.data(), d_W, sizeof(double) * n * batchSize, hipMemcpyDeviceToHost);

    
    for (int i = 0; i < batchSize; i++){
        double temp[n * n];
        double eigenVector[n * n];
        double eigenValue[n];
        for (int j = 0; j < n * n; j++){
            eigenVector[j] = eigenVectors[i * n * n + j];
        }
        for (int j = 0; j < n; j++){
            eigenValue[j] = eigenValues[i * n + j];
        }

        // now perform U^T * V * U
        for (int j = 0; j < n * n; j++){
            temp[j] = 0;
        }
        for (int j = 0; j < n; j++){
            for (int k = 0; k < n; k++){
                for (int l = 0; l < n; l++){
                    temp[j * n + k] += eigenVector[l * n + j] * eigenValue[l] * eigenVector[l * n + k];
                }
            }
        }
        double diff = 0;
        for (int j = 0; j < n * n; j++){
            diff += abs(temp[j] - As[i * n * n + j]);
        }
        if (diff > maximumError){
            maximumError = diff;
        }
    }
    printf("Maximum error: %lf\n", maximumError);
    hipMemcpy(selected_eigen_vectors.data(), d_A + selected_index * n * n, sizeof(double) * n * n, hipMemcpyDeviceToHost);
    hipMemcpy(selected_eigen_values.data(), d_W + selected_index * n, sizeof(double) * n, hipMemcpyDeviceToHost);
    printf("The selected matrix's eigen vectors using cusolver:\n");
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            printf("%lf, ", selected_eigen_vectors[j * n + i]);
        }
        printf("\n");
    }
    printf("\n");
    printf("The selected matrix's eigen values using cusolver:\n");
    for (int i = 0; i < n; i++){
        printf("%lf, ", selected_eigen_values[i]);
    }
    printf("\n\n");


    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_W));
    CHECK_CUDA(hipFree(d_info));
    CHECK_CUDA(hipFree(d_work));
    CHECK_CUSOLVER(hipsolverDnDestroySyevjInfo(syevj_params));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverH));
    CHECK_CUDA(hipDeviceReset());

    return EXIT_SUCCESS;
}