#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "evd.cuh"
#include <vector>
#include <Eigen/Dense>
#include <chrono>

// Error checking macro
#define CHECK_CUDA(call) { \
    const hipError_t error = call; \
    if (error != hipSuccess) { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

#define CHECK_CUSOLVER(call) { \
    const hipsolverStatus_t error = call; \
    if (error != HIPSOLVER_STATUS_SUCCESS) { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("CUSOLVER error code: %d\n", error); \
        exit(1); \
    } \
}

__global__ void generateSymmetricMatrices(double *d_A, int n, unsigned long long seed, int N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n) return;

    // Initialize the random number generator
    hiprandState_t state;
    hiprand_init(seed + tid, tid, 0, &state);

    // Generate the upper triangle of the matrix
    for (int row = 0; row < N; ++row) {
        for (int col = row; col < N; ++col) {
            double randomValue = hiprand_uniform_double(&state) * 2000 - 1000.0;
            d_A[tid * N * N + row * N + col] = randomValue; // Upper triangle
            d_A[tid * N * N + col * N + row] = randomValue; // Mirror to lower triangle
        }
    }

}

template <unsigned int N>
__global__ void evd_global(double* d_A, double* d_e, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n) return;
    evd<N>(d_A + tid * N * N, d_e + tid * N);
}

int main() {
    // ===============================================
    // Setup
    // ===============================================
    hipsolverHandle_t cusolverH = NULL;
    hipsolverSyevjInfo_t syevj_params = NULL;
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // Compute eigenvalues and eigenvectors
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    const unsigned int n = 9; // size of each matrix
    int lda = n;
    int batchSize = 1000000;
    double *d_A = NULL; // Device matrix
    double *d_W = NULL; // Device eigenvalues
    int *d_info = NULL; // info on device
    double *d_work = NULL; // workspace on device
    int lwork = 0; // workspace size
    int info_gpu = 0; // info on host
    hipEvent_t start, stop;
    float milliseconds = 0;
    unsigned int selected_index = 1893;
    std::vector<double> selected_matrix;
    selected_matrix.resize(n * n);
    std::vector<double> selected_eigen_vectors;
    selected_eigen_vectors.resize(n * n);
    std::vector<double> selected_eigen_values;
    selected_eigen_values.resize(n);


    // Initialize cuSolver
    CHECK_CUSOLVER(hipsolverDnCreate(&cusolverH));
    // Allocate memory on device
    CHECK_CUDA(hipMalloc((void**)&d_A, sizeof(double) * lda * n * batchSize));
    CHECK_CUDA(hipMalloc((void**)&d_W, sizeof(double) * n * batchSize));
    CHECK_CUDA(hipMalloc((void**)&d_info, sizeof(int) * batchSize));
    // Setup the execution configuration
    int threadsPerBlock = 32;
    int blocksPerGrid = (batchSize + threadsPerBlock - 1) / threadsPerBlock;
    std::vector<double> eigenVectors;
    std::vector<double> eigenValues;
    eigenVectors.resize(n * n * batchSize);
    eigenValues.resize(n * batchSize);
    double maximumError = 0.0;


    // ===============================================
    // Generate random symmetric matrices
    // ===============================================
    // Seed for the random number generator
    unsigned long long seed = 13;
    // Launch the kernel to generate random symmetric matrices
    generateSymmetricMatrices<<<blocksPerGrid, threadsPerBlock>>>(d_A, batchSize, seed, n);
    hipDeviceSynchronize();


    // ===============================================
    // check and save the matrices
    // ===============================================
    std::vector<double> As;
    As.resize(n * n * batchSize);
    auto startChrono2 = std::chrono::high_resolution_clock::now();
    hipMemcpy(As.data(), d_A, sizeof(double) * n * n * batchSize, hipMemcpyDeviceToHost);
    auto endChrono2 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds2 = endChrono2-startChrono2;
    printf("Time for copying back the original matrix: %f ms\n", elapsed_seconds2.count() * 1000);
    printf("Random matrix generated and stored on host and device\n");
    hipMemcpy(selected_matrix.data(), d_A + selected_index * n * n, sizeof(double) * n * n, hipMemcpyDeviceToHost);

    printf("The selected matrix:\n");
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            printf("%lf, ", selected_matrix[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");


    // ===============================================
    // Eigen value decomposition with our code
    // ===============================================
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    evd_global<n><<<blocksPerGrid, threadsPerBlock>>>(d_A, d_W, batchSize);
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time for 1 execution of our eigen value decomposition: %f ms\n", milliseconds);

    // ===============================================
    // check the error for our solution
    // ===============================================
    hipMemcpy(eigenVectors.data(), d_A, sizeof(double) * n * n * batchSize, hipMemcpyDeviceToHost);
    hipMemcpy(eigenValues.data(), d_W, sizeof(double) * n * batchSize, hipMemcpyDeviceToHost);

    for (int i = 0; i < batchSize; i++){
        double temp[n * n];
        double eigenVector[n * n];
        double eigenValue[n];
        for (int j = 0; j < n * n; j++){
            eigenVector[j] = eigenVectors[i * n * n + j];
        }
        for (int j = 0; j < n; j++){
            eigenValue[j] = eigenValues[i * n + j];
        }

        // now perform U^T * V * U
        for (int j = 0; j < n * n; j++){
            temp[j] = 0;
        }
        for (int j = 0; j < n; j++){
            for (int k = 0; k < n; k++){
                for (int l = 0; l < n; l++){
                    temp[j * n + k] += eigenVector[j * n + l] * eigenValue[l] * eigenVector[k * n + l];
                    if (temp != temp){
                        printf("NAN detected in eigen vector or eigen values\n");
                        exit(1);
                    }
                }
            }
        }
        double diff = 0;
        for (int j = 0; j < n * n; j++){
            diff += abs(temp[j] - As[i * n * n + j]);
        }
        if (diff > maximumError){
            maximumError = diff;
        }
    }
    printf("Maximum error: %lf\n", maximumError);
    maximumError = 0.0;
    

    hipMemcpy(selected_eigen_vectors.data(), d_A + selected_index * n * n, sizeof(double) * n * n, hipMemcpyDeviceToHost);
    hipMemcpy(selected_eigen_values.data(), d_W + selected_index * n, sizeof(double) * n, hipMemcpyDeviceToHost);
    printf("The selected matrix's eigen vectors using our method:\n");
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            printf("%lf, ", selected_eigen_vectors[i * n + j]);
        }
        printf("\n");
    }
    printf("\n");
    printf("The selected matrix's eigen values using our method:\n");
    for (int i = 0; i < n; i++){
        printf("%lf, ", selected_eigen_values[i]);
    }
    printf("\n\n");

    // ===============================================
    // setting up Eigen value decomposition with cuSolver
    // ===============================================
    // first we copy back the original matrix
    auto startChrono1 = std::chrono::high_resolution_clock::now();
    hipMemcpy(d_A, As.data(), sizeof(double) * n * n * batchSize, hipMemcpyHostToDevice);
    auto endChrono1 = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds1 = endChrono1-startChrono1;
    printf("Time for copying back the original matrix: %f ms\n", elapsed_seconds1.count() * 1000);

    // Create syevj parameters
    CHECK_CUSOLVER(hipsolverDnCreateSyevjInfo(&syevj_params));

    // Set syevj parameters
    // For example, setting the tolerance and the maximum number of sweeps:
    CHECK_CUSOLVER(hipsolverDnXsyevjSetTolerance(syevj_params, 1e-6));
    CHECK_CUSOLVER(hipsolverDnXsyevjSetMaxSweeps(syevj_params, 100));

    // Query working space of syevjBatched
    CHECK_CUSOLVER(hipsolverDnDsyevjBatched_bufferSize(
        cusolverH,
        jobz,
        uplo,
        n,
        d_A,
        lda,
        d_W,
        &lwork,
        syevj_params,
        batchSize));

    // Allocate workspace for device
    CHECK_CUDA(hipMalloc((void**)&d_work, sizeof(double) * lwork));
    hipDeviceSynchronize();


    // ===============================================
    // cusolver execute
    // ===============================================
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    // Execute the eigendecomposition
    hipsolverDnDsyevjBatched(
        cusolverH,
        jobz,
        uplo,
        n,
        d_A,
        lda,
        d_W,
        d_work,
        lwork,
        d_info,
        syevj_params,
        batchSize);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time for 1 execution of cusolver eigen value decomposition: %f ms\n", milliseconds);
    // Synchronize and check for errors
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipMemcpy(&info_gpu, d_info, sizeof(int), hipMemcpyDeviceToHost));

    if (info_gpu == 0) {
        printf("Batched syevj execution successful!\n");
    } else {
        printf("Batched syevj execution failed: Info = %d\n", info_gpu);
    }

    // ===============================================
    // check the error for cuSolver solution
    // ===============================================

    hipMemcpy(eigenVectors.data(), d_A, sizeof(double) * n * n * batchSize, hipMemcpyDeviceToHost);
    hipMemcpy(eigenValues.data(), d_W, sizeof(double) * n * batchSize, hipMemcpyDeviceToHost);

    
    for (int i = 0; i < batchSize; i++){
        double temp[n * n];
        double eigenVector[n * n];
        double eigenValue[n];
        for (int j = 0; j < n * n; j++){
            eigenVector[j] = eigenVectors[i * n * n + j];
        }
        for (int j = 0; j < n; j++){
            eigenValue[j] = eigenValues[i * n + j];
        }

        // now perform U^T * V * U
        for (int j = 0; j < n * n; j++){
            temp[j] = 0;
        }
        for (int j = 0; j < n; j++){
            for (int k = 0; k < n; k++){
                for (int l = 0; l < n; l++){
                    temp[j * n + k] += eigenVector[l * n + j] * eigenValue[l] * eigenVector[l * n + k];
                }
            }
        }
        double diff = 0;
        for (int j = 0; j < n * n; j++){
            diff += abs(temp[j] - As[i * n * n + j]);
        }
        if (diff > maximumError){
            maximumError = diff;
        }
    }
    printf("Maximum error: %lf\n", maximumError);
    hipMemcpy(selected_eigen_vectors.data(), d_A + selected_index * n * n, sizeof(double) * n * n, hipMemcpyDeviceToHost);
    hipMemcpy(selected_eigen_values.data(), d_W + selected_index * n, sizeof(double) * n, hipMemcpyDeviceToHost);
    printf("The selected matrix's eigen vectors using cusolver:\n");
    for (int i = 0; i < n; i++){
        for (int j = 0; j < n; j++){
            printf("%lf, ", selected_eigen_vectors[j * n + i]);
        }
        printf("\n");
    }
    printf("\n");
    printf("The selected matrix's eigen values using cusolver:\n");
    for (int i = 0; i < n; i++){
        printf("%lf, ", selected_eigen_values[i]);
    }
    printf("\n\n");


    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_W));
    CHECK_CUDA(hipFree(d_info));
    CHECK_CUDA(hipFree(d_work));
    CHECK_CUSOLVER(hipsolverDnDestroySyevjInfo(syevj_params));
    CHECK_CUSOLVER(hipsolverDnDestroy(cusolverH));
    CHECK_CUDA(hipDeviceReset());
    auto startChrono = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < batchSize; ++i) {
        Eigen::Map<Eigen::Matrix<double, n, n, Eigen::RowMajor>> matrix(As.data() + i * n * n);
    
        // Compute the eigenvalue decomposition
        Eigen::SelfAdjointEigenSolver<Eigen::Matrix<double, n, n>> solver(matrix);
    
        // Eigenvalues
        auto eigenvalues = solver.eigenvalues();
        // Eigenvectors
        auto eigenvectors = solver.eigenvectors();
        // if (i == selected_index){
        //     // print the eigen vectors and eigen values
        //     printf("The selected matrix's eigen vectors using Eigen:\n");
        //     for (int i = 0; i < n; i++){
        //         for (int j = 0; j < n; j++){
        //             printf("%lf, ", eigenvectors(j, i));
        //         }
        //         printf("\n");
        //     }
        //     printf("\n");
        //     printf("The selected matrix's eigen values using Eigen:\n");
        //     for (int i = 0; i < n; i++){
        //         printf("%lf, ", eigenvalues(i));
        //     }
        //     printf("\n\n");
        // }
    }
    auto endChrono = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds = endChrono-startChrono;
    printf("Time for 1 execution of Eigen value decomposition: %f ms\n", elapsed_seconds.count() * 1000);


    return EXIT_SUCCESS;
}